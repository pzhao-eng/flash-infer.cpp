#include "hip/hip_runtime.h"
#include "cuda_utils.h"

void output_as_bin(const char *filename, const char *data, const int n) {
  FILE *fp = fopen(filename, "wb");
  fwrite(data, sizeof(char), n, fp);
  fclose(fp);
}

template <typename kernel_traits>
void run_flash_infer(const FlashInferParams params) {
  constexpr int  kNThreads = kernel_traits::kNThreads;
  constexpr int kBlockM = kernel_traits::kBlockM;
  constexpr int kBlockN = kernel_traits::kBlockN;
  constexpr int kSmemSize = kernel_traits::kSmemSize;
  const int seqlen_q = params.seqlen_q;
  const int seqlen_kv = params.seqlen_kv;
  const int n_heads = params.head_num;

  dim3 block(kNThreads);
  int grid_x = (seqlen_q + kBlockM - 1) / kBlockM;
  dim3 grid(grid_x, n_heads);
  const int n_block_kv = cute::ceil_div(seqlen_kv, kBlockN);
  bool is_even_MN = seqlen_q % kBlockM == 0 && seqlen_kv % kBlockN == 0;
  if (is_even_MN) {
    flash_infer_kernel<kernel_traits, true><<<grid, block, kSmemSize>>>(params, n_block_kv);
  } else {
    flash_infer_kernel<kernel_traits, false><<<grid, block, kSmemSize>>>(params, n_block_kv);
  }
}

template <typename Element>
void prepare_input(FlashInferParams &params) {

  const int seqlen_q = params.seqlen_q;
  const int seqlen_kv = params.seqlen_kv;
  const int n_heads = params.head_num;
  const int head_dims_q = params.head_dims_q;
  const int head_dims_kv = params.head_dims_kv;
  const int dims_q = seqlen_q * n_heads * head_dims_q;
  const int dims_kv = seqlen_kv * n_heads * head_dims_kv;
  const int n_bytes_q = dims_q * sizeof(Element);
  const int n_bytes_kv = dims_kv * sizeof(Element);
  const int n_bytes_output = dims_q * sizeof(Element);
  Element *Q_host = (Element*)malloc(n_bytes_q);
  Element *K_host = (Element*)malloc(n_bytes_kv);
  Element *V_host = (Element*)malloc(n_bytes_kv);
  Element *O_host = (Element*)malloc(n_bytes_output);
  Element *O_host_ref = (Element*)malloc(n_bytes_output);
  void *Q_dev = nullptr;
  void *K_dev = nullptr;
  void *V_dev = nullptr;
  void *O_dev = nullptr;
  hipMalloc(&Q_dev, n_bytes_q);
  hipMalloc(&K_dev, n_bytes_kv);
  hipMalloc(&V_dev, n_bytes_kv);
  hipMalloc(&O_dev, n_bytes_output);

  float softmax_scale = 1.0 / sqrtf(float(head_dims_q));
  float softmax_scale_log2 = softmax_scale * M_LOG2E;
  hipStream_t stream = nullptr;
  hipStreamCreate(&stream);
  
  params.Q_ptr = Q_dev;
  params.K_ptr = K_dev;
  params.V_ptr = V_dev;
  params.output_ptr = O_dev;
  params.output_host = O_host;
  params.stream = stream;
  params.softmax_scale = softmax_scale;
  params.softmax_scale_log2 = softmax_scale_log2;

  // set rand seed
  srand(0);
  for (int i = 0; i < dims_q; i++) {
    // Q_host[i] = static_cast<Element>(i + 1) * 0.1;
    // gen random float number
    float random = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    Q_host[i] = static_cast<Element>(random);
    int index = rand() % dims_q;
    Q_host[index] = 0.0;
  }
  for (int i = 0; i < dims_kv; i++) {
    // K_host[i] = static_cast<Element>(i + 1) * 0.1;
    // V_host[i] = static_cast<Element>(i + 1) * 0.1;
    float random = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    K_host[i] = static_cast<Element>(random);
    V_host[i] = static_cast<Element>(random);
  }

  for (int i = 0; i < 100; i++) {
    int index = rand() % dims_q;
    Q_host[index] = 0.0;
    int index_b = rand() % dims_kv;
    K_host[index_b] = 0.0;
    V_host[index_b] = 0.0;
  }

  hipMemcpy(params.Q_ptr, Q_host, n_bytes_q, hipMemcpyHostToDevice);
  hipMemcpy(params.K_ptr, K_host, n_bytes_kv, hipMemcpyHostToDevice);
  hipMemcpy(params.V_ptr, V_host, n_bytes_kv, hipMemcpyHostToDevice);
  hipMemset(params.output_ptr, 0, n_bytes_output);

  // output as bin file
  output_as_bin("q.bin", (const char*)Q_host, n_bytes_q);
  output_as_bin("k.bin", (const char*)K_host, n_bytes_kv);
  output_as_bin("v.bin", (const char*)V_host, n_bytes_kv);

  printf("input data sample:\n");
  printf("Q_host[0] A[0]: %f A[1]: %f\n", float(Q_host[0]), float(Q_host[1]));
  printf("K_host[0] B[0]: %f B[1]: %f\n", float(K_host[0]), float(K_host[1]));
  printf("V_host[0] V[0]: %f V[1]: %f\n", float(V_host[0]), float(V_host[1]));
  free(Q_host);
  free(K_host);
  free(V_host);
}

template void prepare_input<cutlass::half_t>(FlashInferParams &params);
template void run_flash_infer<FlashInferTraits<32, 128, 128, 4, cutlass::half_t>>(const FlashInferParams params);